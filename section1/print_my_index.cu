#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_device_set.h"

#include <stdio.h>


__global__ void print_my_index()
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	printf("my id :%d , block_id :%d \n",tid,bid);
}


int main()
{
	cudaDeviceInit();
	
	printf("hello from main \n");
	print_my_index << <4, 10 >> > ();
	hipDeviceSynchronize();
	return 0;
}