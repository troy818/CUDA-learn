#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>       // helper for CUDA Error handling and initialization

/**
 * Naive erosion kernel with each thread processing a square area.
 */
__global__ void NaiveErosionKernel(int * src, int * dst, int width, int height, int radio) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y >= height || x >= width) {
        return;
    }
    unsigned int start_i = max(y - radio, 0);
    unsigned int end_i = min(height - 1, y + radio);
    unsigned int start_j = max(x - radio, 0);
    unsigned int end_j = min(width - 1, x + radio);
    int value = 255;
    for (int i = start_i; i <= end_i; i++) {
        for (int j = start_j; j <= end_j; j++) {
            value = min(value, src[i * width + j]);
        }
    }
    dst[y * width + x] = value;
}

void NaiveErosion(int * src, int * dst, int width, int height, int radio) {
    dim3 block(32, 32);
    dim3 grid(ceil((float)width / block.x), ceil((float)height / block.y));
    NaiveErosionKernel<<<grid,block>>>(src, dst, width, height, radio);
    hipError_t cudaerr = hipDeviceSynchronize();
}

/**
 * Two steps erosion using separable filters
 */
__global__ void ErosionStep2(int * src, int * dst, int width, int height, int radio) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y >= height || x >= width) {
        return;
    }
    unsigned int start_i = max(y - radio, 0);
    unsigned int end_i = min(height - 1, y + radio);
    int value = 255;
    for (int i = start_i; i <= end_i; i++) {
        value = min(value, src[i * width + x]);
    }
    dst[y * width + x] = value;
}

__global__ void ErosionStep1(int * src, int * dst, int width, int height, int radio) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y >= height || x >= width) {
        return;
    }
    unsigned int start_j = max(x - radio, 0);
    unsigned int end_j = min(width - 1, x + radio);
    int value = 255;
    for (int j = start_j; j <= end_j; j++) {
        value = min(value, src[y * width + j]);
    }
    dst[y * width + x] = value;
}

void ErosionTwoSteps(int * src, int * dst, int * temp, int width, int height, int radio) {
    dim3 block(16, 16);
    dim3 grid(ceil((float)width / block.x), ceil((float)height / block.y));
    ErosionStep1<<<grid,block>>>(src, temp, width, height, radio);
    hipError_t cudaerr = hipDeviceSynchronize();
    ErosionStep2<<<grid,block>>>(temp, dst, width, height, radio);
    cudaerr = hipDeviceSynchronize();
}


/**
 * Two steps erosion using separable filters with shared memory.
 */
__global__ void ErosionSharedStep2(int * src, int *src_src, int * dst, int radio, int width, int height, int tile_w, int tile_h) {
    extern __shared__ int smem[];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int x = bx * tile_w + tx;
    int y = by * tile_h + ty - radio;
    smem[ty * blockDim.x + tx] = 255;
    __syncthreads();
    if (x >= width || y < 0 || y >= height) {
        return;
    }
    smem[ty * blockDim.x + tx] = src[y * width + x];
    __syncthreads();
    if (y < (by * tile_h) || y >= ((by + 1) * tile_h)) {
        return;
    }
    int * smem_thread = &smem[(ty - radio) * blockDim.x + tx];
    int val = smem_thread[0];
    for (int yy = 1; yy <= 2 * radio; yy++) {
        val = min(val, smem_thread[yy * blockDim.x]);
    }
    dst[y * width + x] = val;
}

__global__ void ErosionSharedStep1(int * src, int * dst, int radio, int width, int height, int tile_w, int tile_h) {
    extern __shared__ int smem[];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int x = bx * tile_w + tx - radio;
    int y = by * tile_h + ty;
    smem[ty * blockDim.x + tx] = 255;
    __syncthreads();
    if (x < 0 || x >= width || y >= height) {
        return;
    }
    smem[ty * blockDim.x + tx] = src[y * width + x];
    __syncthreads();
    if (x < (bx * tile_w) || x >= ((bx + 1) * tile_w)) {
        return;
    }
    int * smem_thread = &smem[ty * blockDim.x + tx - radio];
    int val = smem_thread[0];
    for (int xx = 1; xx <= 2 * radio; xx++) {
        val = min(val, smem_thread[xx]);
    }
    dst[y * width + x] = val;
}

void ErosionTwoStepsShared(int * src, int * dst, int * temp, int width, int height, int radio) {
    int tile_w = 640;
    int tile_h = 1;
    dim3 block2(tile_w + (2 * radio), tile_h);
    dim3 grid2(ceil((float)width / tile_w), ceil((float)height / tile_h));
    ErosionSharedStep1<<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, radio, width, height, tile_w, tile_h);
    hipError_t cudaerr = hipDeviceSynchronize();
    tile_w = 8;
    tile_h = 64;
    dim3 block3(tile_w, tile_h + (2 * radio));
    dim3 grid3(ceil((float)width / tile_w), ceil((float)height / tile_h));
    ErosionSharedStep2<<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, src, dst, radio, width, height, tile_w, tile_h);
    cudaerr = hipDeviceSynchronize();
}

/*
template<const int radio> __global__ void ErosionTemplateSharedStep2(int * src, int * dst, int width, int height, int tile_w, int tile_h) {
    extern __shared__ int smem[];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int x = bx * tile_w + tx;
    int y = by * tile_h + ty - radio;
    smem[ty * blockDim.x + tx] = 255;
    __syncthreads();
    if (x >= width || y < 0 || y >= height) {
        return;
    }
    smem[ty * blockDim.x + tx] = src[y * width + x];
    __syncthreads();
    if (y < (by * tile_h) || y >= ((by + 1) * tile_h)) {
        return;
    }
    int * smem_thread = &smem[(ty - radio) * blockDim.x + tx];
    int val = smem_thread[0];
#pragma unroll
    for (int yy = 1; yy <= 2 * radio; yy++) {
        val = min(val, smem_thread[yy * blockDim.x]);
    }
    dst[y * width + x] = val;
}

template<const int radio> __global__ void ErosionTemplateSharedStep1(int * src, int * dst, int width, int height, int tile_w, int tile_h) {
    extern __shared__ int smem[];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int x = bx * tile_w + tx - radio;
    int y = by * tile_h + ty;
    smem[ty * blockDim.x + tx] = 255;
    __syncthreads();
    if (x < 0 || x >= width || y >= height) {
        return;
    }
    smem[ty * blockDim.x + tx] = src[y * width + x];
    __syncthreads();
    if (x < (bx * tile_w) || x >= ((bx + 1) * tile_w)) {
        return;
    }
    int * smem_thread = &smem[ty * blockDim.x + tx - radio];
    int val = smem_thread[0];
#pragma unroll
    for (int xx = 1; xx <= 2 * radio; xx++) {
        val = min(val, smem_thread[xx]);
    }
    dst[y * width + x] = val;
}

void ErosionTemplateSharedTwoSteps(int * src, int * dst, int * temp, int width, int height, int radio) {
    int tile_w1 = 256, tile_h1 = 1;
    dim3 block2(tile_w1 + (2 * radio), tile_h1);
    dim3 grid2(ceil((float)width / tile_w1), ceil((float)height / tile_h1));
    int tile_w2 = 4, tile_h2 = 64;
    dim3 block3(tile_w2, tile_h2 + (2 * radio));
    dim3 grid3(ceil((float)width / tile_w2), ceil((float)height / tile_h2));
    switch (radio) {
        case 1:
            ErosionTemplateSharedStep1<1><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<1><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 2:
            ErosionTemplateSharedStep1<2><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<2><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 3:
            ErosionTemplateSharedStep1<3><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<3><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 4:
            ErosionTemplateSharedStep1<4><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<4><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 5:
            ErosionTemplateSharedStep1<5><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<5><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 6:
            ErosionTemplateSharedStep1<6><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<6><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 7:
            ErosionTemplateSharedStep1<7><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<7><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 8:
            ErosionTemplateSharedStep1<8><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<8><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 9:
            ErosionTemplateSharedStep1<9><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<9><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 10:
            ErosionTemplateSharedStep1<10><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<10><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 11:
            ErosionTemplateSharedStep1<11><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<11><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 12:
            ErosionTemplateSharedStep1<12><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<12><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 13:
            ErosionTemplateSharedStep1<13><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<13><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 14:
            ErosionTemplateSharedStep1<14><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<14><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
        case 15:
            ErosionTemplateSharedStep1<15><<<grid2,block2,block2.y*block2.x*sizeof(int)>>>(src, temp, width, height, tile_w1, tile_h1);
            checkCudaErrors(hipDeviceSynchronize());
            ErosionTemplateSharedStep2<15><<<grid3,block3,block3.y*block3.x*sizeof(int)>>>(temp, dst, width, height, tile_w2, tile_h2);
            break;
    }
    hipError_t cudaerr = hipDeviceSynchronize();
}
*/